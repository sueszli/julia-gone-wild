#include <iostream>
#include "kernel.hpp"
#include <hip/hip_runtime.h>

extern int global_block_x, global_block_y;

void julia_kernel(float *julia_set, Complex c, float scale, int res_x, int res_y, int max_iter, float max_mag, float x_scale, float y_scale) {

    // compute a good default block size
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // thread safety
    if (x >= res_x || y >= res_y) {
        return;
    }

    // calculate coordinates
    float scaledX = scale * x_scale * (float)(x - res_x / 2) / (res_x / 2);
    float scaledY = scale * y_scale * (float)(y - res_y / 2) / (res_y / 2);

    Complex z(scaledX, scaledY);

    int i = 0;
    for(i = 0; i < max_iter; i++) {
        z = z * z + c;
        if(z.magnitude2() > max_mag) {
            break;
        }
    }

    // calculate shade for julia set value
    float juliaShade = ((float)i) / max_iter;
    julia_set[y * res_x + x] = juliaShade; // note the swapped indices due to row-major order in memory
}

void launch_julia_kernel(float *d_julia_set, Complex c, float scale, int res_x, int res_y, int max_iter, float max_mag, float x_scale, float y_scale) {
    dim3 blockSize(16, 16); // example block size -> must be adjusted based on the GPU specs
    dim3 gridSize((res_x + blockSize.x - 1) / blockSize.x, (res_y + blockSize.y - 1) / blockSize.y);

    julia_kernel<<<gridSize, blockSize>>>(d_julia_set, c, scale, res_x, res_y, max_iter, max_mag, x_scale, y_scale);
}
